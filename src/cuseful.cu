#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<string.h>
#include<hipblas.h>
#include<R.h>
#include<Rinternals.h>

#include"cuseful.h"

#define HALF RAND_MAX/2

int hasCudaError(const char * msg) {
	hipError_t err = hipGetLastError();
	if(hipSuccess != err)
		error("cuda error : %s : %s\n", msg, hipGetErrorString(err));
	return 0;
}

void checkCudaError(const char * msg) {
	hipError_t err = hipGetLastError();
	if(hipSuccess != err) {
		if(msg != NULL)
			warning(msg);
		error(hipGetErrorString(err));
	}
}

char * cublasGetErrorString(hipblasStatus_t err)
{
	switch(err) {
		case HIPBLAS_STATUS_SUCCESS :
			return "operation completed successfully";
		case HIPBLAS_STATUS_NOT_INITIALIZED :
			return "CUBLAS library not initialized";
		case HIPBLAS_STATUS_ALLOC_FAILED :
			return "resource allocation failed";
		case HIPBLAS_STATUS_INVALID_VALUE :
			return "unsupported numerical value was passed to function";
		case HIPBLAS_STATUS_ARCH_MISMATCH :
			return "function requires an architectural feature absent from \
			the architecture of the device";
		case HIPBLAS_STATUS_MAPPING_ERROR :
			return "access to GPU memory space failed";
		case HIPBLAS_STATUS_EXECUTION_FAILED :
			return "GPU program failed to execute";
		case HIPBLAS_STATUS_INTERNAL_ERROR :
			return "an internal CUBLAS operation failed";
		default :
			return "unknown error type";
	}
}

void checkCublasError(const char * msg)
{
	hipblasStatus_t err = cublasGetError();
	if(err != HIPBLAS_STATUS_SUCCESS)
		error("cublas error : %s : %s\n", msg, cublasGetErrorString(err));
}

int hasCublasError(const char * msg)
{
	hipblasStatus_t err = cublasGetError();
	if(err != HIPBLAS_STATUS_SUCCESS)
		error("cublas error : %s : %s\n", msg, cublasGetErrorString(err));
	return 0;
}
