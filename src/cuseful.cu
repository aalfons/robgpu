#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<string.h>
#include<hipblas.h>
#include<R.h>
#include<Rinternals.h>

#include"cuseful.h"

#define HALF RAND_MAX/2

void fatal(const char * msg)
{
	error(msg);
}

void getComputeNumber(int * major, int * minor)
{
	int currentDevice = 0;
	struct hipDeviceProp_t dProps;

	hipGetDevice(&currentDevice);
	hipGetDeviceProperties(&dProps, currentDevice);

	*major = dProps.major;
	*minor = dProps.minor;
}

void checkDoubleCapable(const char * failMsg)
{
	int major, minor;
	major = minor = 0;
	getComputeNumber(&major, &minor);
	if((major < 1) || ((major == 1) && (minor < 3)))
		error(failMsg);
}

float * getMatFromFile(int rows, int cols, const char * fn)
{
	FILE * matFile;
	matFile = fopen(fn, "r");
	if(matFile == NULL)
		error("unable to open file %s", fn);
	float * mat = Calloc(rows*cols, float);
	int i, j, err;
	for(i = 0; i < rows; i++) {
		for(j = 0; j < cols; j++) {
			err = fscanf(matFile, " %f ", mat+i+j*rows);
			if(err == EOF)
				error("file %s incorrect: formatting or size", fn);
		}
		fscanf(matFile, " \n ");
	}
	fclose(matFile);
	return mat;
}

char * getTime() {
	time_t curtime;
	struct tm *loctime;
	curtime = time(NULL);
	loctime = localtime(&curtime);

	return asctime(loctime);
}

void printVect(int n, const float * vect, const char * msg) {
	if(msg != NULL) Rprintf(msg);
	for(int i = 0; i < n; i++) {
		Rprintf("%6.4f, ", vect[i]);
		if((i+1)%10 == 0) Rprintf("\n");
	}
	if(n%10 != 0) Rprintf("\n");
	if(msg != NULL) Rprintf("----------\n");
}

void printMat(int rows, int cols, const float * mat, const char * msg) {
	int i;
	if(msg != NULL) Rprintf(msg);
	for(i = 0; i < rows; i++)
		printVect(cols, mat+i*cols, NULL);
	if(msg != NULL) Rprintf("----------\n");
}

void getRandVect(float * vect, size_t n) {
	srand(time(0));
	for(size_t i = 0; i < n; i++)
		vect[i] = ((float)rand())/((float)RAND_MAX);
}

void getBernVect(float * vect, size_t n) {
	srand(time(0));
	for(size_t i = 0; i < n; i++) {
		if(rand() <= HALF) vect[i] = 1.f;
		else vect[i] = 0.f;
	}
}

int hasCudaError(const char * msg) {
	hipError_t err = hipGetLastError();
	if(hipSuccess != err)
		error("cuda error : %s : %s\n", msg, hipGetErrorString(err));
	return 0;
}

void checkCudaError(const char * msg) {
	hipError_t err = hipGetLastError();
	if(hipSuccess != err) {
		if(msg != NULL)
			warning(msg);
		error(hipGetErrorString(err));
	}
}

char * cublasGetErrorString(hipblasStatus_t err)
{
	switch(err) {
		case HIPBLAS_STATUS_SUCCESS :
			return "operation completed successfully";
		case HIPBLAS_STATUS_NOT_INITIALIZED :
			return "CUBLAS library not initialized";
		case HIPBLAS_STATUS_ALLOC_FAILED :
			return "resource allocation failed";
		case HIPBLAS_STATUS_INVALID_VALUE :
			return "unsupported numerical value was passed to function";
		case HIPBLAS_STATUS_ARCH_MISMATCH :
			return "function requires an architectural feature absent from \
			the architecture of the device";
		case HIPBLAS_STATUS_MAPPING_ERROR :
			return "access to GPU memory space failed";
		case HIPBLAS_STATUS_EXECUTION_FAILED :
			return "GPU program failed to execute";
		case HIPBLAS_STATUS_INTERNAL_ERROR :
			return "an internal CUBLAS operation failed";
		default :
			return "unknown error type";
	}
}

void checkCublasError(const char * msg)
{
	hipblasStatus_t err = cublasGetError();
	if(err != HIPBLAS_STATUS_SUCCESS)
		error("cublas error : %s : %s\n", msg, cublasGetErrorString(err));
}

int hasCublasError(const char * msg)
{
	hipblasStatus_t err = cublasGetError();
	if(err != HIPBLAS_STATUS_SUCCESS)
		error("cublas error : %s : %s\n", msg, cublasGetErrorString(err));
	return 0;
}
